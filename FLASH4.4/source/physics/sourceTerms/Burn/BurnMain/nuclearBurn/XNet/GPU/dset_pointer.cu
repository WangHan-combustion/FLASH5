#include "hip/hip_runtime.h"
#include "dbatched.h"

/******************************************************************************/
__global__ void kernel_dset_pointer(
    double **output_array,
    double *input,
    int lda,
    int row, int column, 
    int batch_offset)
{
    output_array[blockIdx.x] =  input + blockIdx.x * batch_offset + row + column * lda;
    //printf("==> kernel_set_pointer input_array %p output_array %p  \n",input+ blockIdx.x * batch_offset,output_array[blockIdx.x]);
}


/***************************************************************************//**
    Purpose
    -------

    convert consecutive stored variable to array stored
    for example the size  of A is N*batchCount; N is the size of A(batch_offset)
    change into dA_array[0] dA_array[1],... dA_array[batchCount-1], where the size of each dA_array[i] is N
    
    Arguments
    ----------

    @param[out]
    output_array  Array of pointers, dimension (batchCount).
             Each is a DOUBLE PRECISION array A of DIMENSION ( lda, column ) on the GPU
   
    @param[in]
    input      DOUBLE PRECISION array of dimension ( LDDA, N*batchCount ) on the GPU.


    @param[in]
    lda    INTEGER
            LDA specifies the leading dimension of A.

    @param[in]
    row       INTEGER
            On entry, row specifies the number of rows of the matrix A.

    @param[in]
    column       INTEGER
            On entry, column specifies the number of columns of the matrix A

    @param[in]
    batch_offset  INTEGER
                The starting pointer of each matrix A in input arrray

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    stream   hipStream_t
             stream to execute in.
*******************************************************************************/
extern "C"
void dset_pointer(
    double **output_array,
    double *input,
    int lda,
    int row, int column, 
    int batch_offset,
    int batchCount, 
    hipStream_t stream)
{
    kernel_dset_pointer
        <<< batchCount, 1, 0, stream >>>
        (output_array, input, lda,  row, column, batch_offset);
}


/******************************************************************************/
__global__ void zdisplace_pointers_kernel(double **output_array,
               double **input_array, int lda,
               int row, int column)
{
    double *inpt = input_array[blockIdx.x];
    output_array[blockIdx.x] = &inpt[row + column * lda];
}

/***************************************************************************//**
    Purpose
    -------

    compute the offset for all the matrices and save the displacment of the new pointer on output_array.
    input_array contains the pointers to the initial position.
    output_array[i] = input_array[i] + row + lda * column; 
    
    Arguments
    ----------

    @param[out]
    output_array    Array of pointers, dimension (batchCount).
             Each pointer points to the new displacement of array A in input_array on the GPU
   
    @param[in]
    input_array     Array of pointers, dimension (batchCount).
             Each is a DOUBLE PRECISION array A of DIMENSION ( lda, column ) on the GPU

    @param[in]
    lda    INTEGER
            LDA specifies the leading dimension of A.

    @param[in]
    row       INTEGER
            On entry, row specifies the number of rows of the matrix A.

    @param[in]
    column       INTEGER
            On entry, column specifies the number of columns of the matrix A

    @param[in]
    batch_offset  INTEGER
                The starting pointer of each matrix A in input arrray

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    stream   hipStream_t
             stream to execute in.
*******************************************************************************/
extern "C"
void magma_ddisplace_pointers(double **output_array,
               double **input_array, int lda,
               int row, int column, 
               int batchCount, hipStream_t stream)
{
    zdisplace_pointers_kernel
        <<< batchCount, 1, 0, stream >>>
        (output_array, input_array, lda, row, column);
}
